#include "hip/hip_runtime.h"

#include <chrono>
#include <iostream>
#include <stdexcept>
#include <GL/glew.h>

#include "program.h"
#include "gpu_texture.cuh"
#include "mandelbrot_kernel.cuh"

namespace FastMandelbrot
{

    program::program(unsigned int width, unsigned int height)
        : _width{width}, _height{height}
    {
        SDL_SetHint(SDL_HINT_NO_SIGNAL_HANDLERS, "1");
        SDL_Init(SDL_INIT_VIDEO);

        _window = SDL_CreateWindow(
            "Fast Mandelbrot", 0, 0,
            width, height,
            SDL_WINDOW_OPENGL | SDL_WINDOW_RESIZABLE);

        // Hide the windows until the rendering is not ready to start
        SDL_HideWindow(_window);

        _gl_context = SDL_GL_CreateContext(_window);

        GLenum glew_error = glewInit();
        if (glew_error != GLEW_OK)
        {
            throw std::runtime_error(reinterpret_cast<const char *>(glewGetErrorString(glew_error)));
        }

        SDL_GL_SetAttribute(SDL_GL_DOUBLEBUFFER, 1);
        SDL_GL_SetAttribute(SDL_GL_DEPTH_SIZE, 24);

        // Create openGL texture to render into
        glGenTextures(1, &_texture_id);
        glBindTexture(GL_TEXTURE_2D, _texture_id);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glEnable(GL_TEXTURE_2D);
    }

    program::~program() noexcept
    {
        glDeleteTextures(1, &_texture_id);
        SDL_GL_DeleteContext(_gl_context);
        SDL_DestroyWindow(_window);
    }

    void program::run()
    {
        registered_texture texture{_texture_id, _width, _height};

        SDL_ShowWindow(_window);
        _update_size();

        while (_handle_events(texture));

        SDL_HideWindow(_window);
    }

    bool program::_handle_key_down(SDL_Keysym key)
    {
        switch (key.sym)
        {
            case SDLK_KP_PLUS:
                _step_count *= 2;
                break;

            case SDLK_KP_MINUS:
                _step_count /= 2;
                if (_step_count < 2) _step_count = 1;
                break;

            case SDLK_RETURN:
                break;

            case SDLK_SPACE:
                break;

            default:
                return false;
        }

        std::cout << "step count = " << _step_count << std::endl;
        return true;
    }

    bool program::_handle_mouse_wheel(bool up)
    {
        constexpr auto factor = 1.1f;
        const auto new_size = _size * (up ? (1.f / factor) : factor);
        const auto origin_offset = (_size - new_size) / 2.f;

        _size = new_size;
        _origin_x += origin_offset;
        _origin_y += origin_offset;

        return true;
    }

    bool program::_handle_mouse_drag(int xrel, int yrel)
    {
        const auto unit_per_pixel = _size / static_cast<float>(_width);
        _origin_x -= unit_per_pixel * static_cast<float>(xrel);
        _origin_y += unit_per_pixel * static_cast<float>(yrel);
        return true;
    }

    bool program::_handle_events(registered_texture& texture)
    {
        SDL_Event event;
        bool redraw = false;

        while (SDL_PollEvent(&event)) {
            switch (event.type)
            {
                case SDL_KEYDOWN:
                    redraw = _handle_key_down(event.key.keysym);
                    break;
                case SDL_MOUSEWHEEL:
                    redraw = _handle_mouse_wheel(event.wheel.y > 0);
                    break;
                case SDL_MOUSEMOTION:
                    if (_drag)
                        redraw = _handle_mouse_drag(event.motion.xrel, event.motion.yrel);
                    break;
                case SDL_MOUSEBUTTONDOWN:
                    if (event.button.button == SDL_BUTTON_LEFT) _drag = true;
                    break;
                case SDL_MOUSEBUTTONUP:
                    if (event.button.button == SDL_BUTTON_LEFT) _drag = false;
                    break;
                case SDL_WINDOWEVENT:
                    if (event.window.event == SDL_WINDOWEVENT_RESIZED)
                        _update_size();
                    else if (event.window.event == SDL_WINDOWEVENT_EXPOSED)
                        redraw = true;
                    break;
                case SDL_QUIT:
                    return false;
            }
        }

        if (redraw)
        {
            using namespace std::chrono;

            const auto start = steady_clock::now();
            _render_frame(texture);
            const auto end = steady_clock::now();
            const auto duration = duration_cast<microseconds>(end - start).count();
            const auto fps = static_cast<int>(1.E6 / static_cast<double>(duration));
            std::cout << fps << " fps (" << (duration / 1000u) << " ms)\n";
            _draw_texture();
        }

        return true;
    }

    void program::_draw_texture()
    {
        // Draw a quad with the texture on it (ugly old shoold open gl...)
        glClearColor(0.f, 0.f, 0.f, 0.f);
        glClear(GL_COLOR_BUFFER_BIT);

        glBindTexture(GL_TEXTURE_2D, _texture_id);

        glBegin(GL_QUADS);

        glVertex2i(0, 0);
        glTexCoord2i(1, 0);

        glVertex2i(1, 0);
        glTexCoord2i(1, 1);

        glVertex2i(1, 1);
        glTexCoord2i(0, 1);

        glVertex2i(0, 1);
        glTexCoord2i(0, 0);

        glEnd();

        // double buffering
        SDL_GL_SwapWindow(_window);
    }

    void program::_update_size()
    {
        int width = 0;
        int height = 0;
        SDL_GetWindowSize(_window, &width, &height);
        glViewport(0, 0, width, height);
        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
        glOrtho(0.f, 1.f, 0.f, 1.f, -1.f, 1.f);
    }

    void program::_render_frame(registered_texture& texture)
    {
        const auto origin = float2{_origin_x, _origin_y};
        auto mapped_surface = texture.get_mapped_surface();
        call_mandelbrot_kernel(mapped_surface.surface(), _width, _height, origin, _size, _step_count);
    }
}
